#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


extern  __shared__  float sdata[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
    extern __shared__ float s_data[];
    //Write this kernel to achieve the same output as the provided k0, but you will have to use
    // shared memory.
}

