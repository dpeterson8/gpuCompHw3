#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


extern  __shared__  float sdata[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
    extern __shared__ float s_data[];
    int s_rowwidth = blockDim.x + 2;
    //Write this kernel to achieve the same output as the provided k0, but you will have to use
    // shared memory.
    
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int x = ix + 1;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;
    int y = iy + 1;
    int idx = y * floatpitch + x;

    if( y >= width - 1|| x >= width - 1 || y < 1 || x < 1 ) return;

    s_data[threadIdx.x + 1] = g_dataA[idx - floatpitch];
    s_data[threadIdx.x + 1 + s_rowwidth] = g_dataA[idx];
    s_data[threadIdx.x + 1 + (2 * s_rowwidth)] = g_dataA[idx + floatpitch];


    if(threadIdx.x == 0) {
        s_data[0] = g_dataA[idx - floatpitch - 1];
        s_data[s_rowwidth] = g_dataA[idx - 1];
        s_data[s_rowwidth * 2] = g_dataA[idx + floatpitch - 1];

    } else if (threadIdx.x == blockDim.x - 1 || x + 2 == width) {
        s_data[threadIdx.x + 2] = g_dataA[idx - floatpitch + 1];
        s_data[s_rowwidth + threadIdx.x + 2] = g_dataA[idx + 1];
        s_data[s_rowwidth * 2 + threadIdx.x + 2] = g_dataA[idx + floatpitch + 1];
    }

    // }

    __syncthreads();
    

    int threadId = threadIdx.x + 1;


    g_dataB[idx] = (
        0.2f * s_data[s_rowwidth + threadId] +               //itselfmake
        0.1f * s_data[s_rowwidth + threadId - 1] +       //W
        0.1f * s_data[s_rowwidth + threadId + 1] +      //E
        0.1f * s_data[threadId + 1] +       //NE
        0.1f * s_data[threadId] +       //N
        0.1f * s_data[s_rowwidth * 2 + threadId + 1] +       //SE
        0.1f * s_data[s_rowwidth * 2 + threadId ] +       //S
        0.1f * s_data[s_rowwidth * 2 + threadId - 1] +       //SW
        0.1f * s_data[threadId - 1]         //NW
    )  * 0.95f;

    // __syncthreads();

    // g_dataB[idx] = s_data[(s_rowwidth * 4) + threadId];

    //   g_dataB[idx] = s_data[s_rowwidth + threadId + 1];



    //g_dataB[idx] = s_data[blockDim.x * 3 + x];

}

